#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDA
	#include <hip/hip_runtime_api.h>
	#include <hiprand.h>
	#include <hiprand/hiprand_kernel.h>

	#define NTHREADS (XR*YR/(2*PATCHSIZE*PATCHSIZE))
	hiprandGenerator_t gen;
	float *randnum;
	int *dGrid;
#else
	#define __device__
	#define __host__
#endif

#define BASES 5
#define NEIGHBORS 9

#define XR 320
#define YR 80
#define PATCHSIZE 5
#define RNC 4

#define TEMP 0.5

int colors[3*BASES];
int *cGrid;
int *dN;

class Params
{
	public:
		float *W;
};

Params P;

void setParams()
{
	float *lW = (float*)malloc(BASES*BASES*NEIGHBORS*sizeof(float));
	
	memset(colors,0,sizeof(int)*3*BASES);
	colors[3+0] = 255; colors[3+1] = 0; colors[3+2] = 0;
	colors[6+0] = 0; colors[6+1] = 255; colors[6+2] = 0;
	colors[9+0] = 0; colors[9+1] = 0; colors[9+2] = 255;
	
	for (int i=4;i<BASES;i++)
		for (int j=0;j<3;j++)
			colors[3*i+j] = 64+rand()%180;
			
	for (int i=0;i<BASES*BASES*NEIGHBORS;i++)
		lW[i] = (rand()%2000001-1000000.0)/1000000.0;
	
	for (int i=0;i<BASES;i++)
		for (int j=0;j<NEIGHBORS;j++)
			lW[j + 0*NEIGHBORS + i*BASES*NEIGHBORS] = 0;
			
#ifdef CUDA
	hipMemcpy(P.W,lW,sizeof(float)*BASES*BASES*NEIGHBORS,hipMemcpyHostToDevice);
#else
	memcpy(P.W,lW,sizeof(float)*BASES*BASES*NEIGHBORS);
#endif

	free(lW);
}

__device__ __host__ int MapX(int x)
{
	return min(max(x,0),XR-1);
}

__device__ __host__ int MapY(int x)
{
	return min(max(x,0),YR-1);
}

__device__ __host__ void updateNeighbor(int x, int y, int *Grid, int *dN)
{
	int xm,ym,xm2,ym2;
	int ofs = (x+y*XR)*BASES;
	
	for (int i=0;i<BASES;i++)
		dN[i+ofs]=0;
		
	for (ym=y-1;ym<=y+1;ym++)
		for (xm=x-1;xm<=x+1;xm++)
		{
			xm2=MapX(xm);
			ym2=MapY(ym);
					
			dN[ofs + Grid[xm2+ym2*XR]] += 1;
		}
}

#ifdef CUDA
	__global__ void calcNeighbors(int *Grid, int *dN)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x; // standard...
		int xm2,ym2;
		
		if (idx<XR*YR)
		{
			int x=idx%XR;
			int y=idx/XR;
			
			updateNeighbor(x,y,Grid,dN);
		}
	}				
#else
	void calcNeighbors(int *Grid, int *dN)
	{
		for (int y=0;y<YR;y++)
			for (int x=0;x<XR;x++)
				updateNeighbor(x,y,Grid,dN);
	}
#endif

void setNeighbors()
{
#ifdef CUDA
	int block_size = 32;
	int n_blocks2 = (XR*YR)/block_size + ((XR*YR)%block_size == 0 ? 0 : 1);

	calcNeighbors <<< n_blocks2, block_size >>> (dGrid, dN);
#else
	calcNeighbors(cGrid,dN);
#endif
}

__device__ __host__ float getLocalEnergy(int minx, int miny, int maxx, int maxy, int *Grid, Params P, int *dN)
{
	float E=0;
	int x,y,xm,ym,ofs,loc,i;
	
	for (y=miny;y<=maxy;y++)
		for (x=minx;x<=maxx;x++)
		{
			xm=MapX(x);
			ym=MapY(y);
			ofs = (xm+ym*XR)*BASES;
			loc = Grid[xm+ym*XR]; 
			
			for (i=0;i<BASES;i++)
				E += P.W[dN[ofs + i] - (loc==i) + loc*NEIGHBORS + i*BASES*NEIGHBORS];			
		}
	return E;
}

__device__ __host__ void Swap(int x1, int y1, int x2, int y2, int *Grid, int *dN, int src, int sink)
{
	int xm3,ym3;
	
	for (int y3=y1-1;y3<=y1+1;y3++)
		for (int x3=x1-1;x3<=x1+1;x3++)
		{
			xm3 = MapX(x3); 
			ym3 = MapY(y3);
			
			dN[(xm3+ym3*XR)*BASES+sink] += 1;
			dN[(xm3+ym3*XR)*BASES+src] -= 1;
		}
			
	for (int y3=y2-1;y3<=y2+1;y3++)
		for (int x3=x2-1;x3<=x2+1;x3++)
		{
			xm3 = MapX(x3); 
			ym3 = MapY(y3);
			
			dN[(xm3+ym3*XR)*BASES+sink] -= 1;
			dN[(xm3+ym3*XR)*BASES+src] += 1;
		}

	Grid[x1+y1*XR] = sink;
	Grid[x2+y2*XR] = src;	
}

__device__ __host__ void tryMove(int x1, int y1, int x2, int y2, int *Grid, int *dN, Params P, float r)
{
	int mnx = min(x1,x2), mny = min(y1,y2), mxx = max(x1,x2), mxy = max(y1,y2);
	x2 = MapX(x2);
	y2 = MapY(y2);
	
	int src = Grid[x1+y1*XR];
	int sink = Grid[x2+y2*XR];
			
	if (src != sink)
	{					
		float E1 = getLocalEnergy(mnx-1,mny-1,mxx+1,mxy+1,Grid,P,dN);
		
		Swap(x1,y1,x2,y2,Grid,dN,src,sink);
		
		float E2 = getLocalEnergy(mnx-1,mny-1,mxx+1,mxy+1,Grid,P,dN);
		
		if (E2>E1)
		{
			if (r>exp(-(E2-E1)/TEMP))
			{
				Swap(x1,y1,x2,y2,Grid,dN,sink,src);
			}
		}
	}
}

#ifdef CUDA
__global__ void PatchKernel(int *Grid, int ox, int oy, int flip, Params P, float *rnum, int iter, int *dN)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // standard...
	
	if (idx<NTHREADS)
	{
		int x1,y1,d,x2,y2,xp,yp;
		int ofs = idx*RNC+iter*RNC*NTHREADS;
		
		yp = idx/(XR/(2*PATCHSIZE));
		xp = 2*(idx%(XR/(2*PATCHSIZE)))+((yp+flip)%2);
		
		x1 = MapX(ox+(int)floor(rnum[ofs+0]*(PATCHSIZE-2)+1) + xp*PATCHSIZE);
		y1 = MapY(oy+(int)floor(rnum[ofs+1]*(PATCHSIZE-2)+1) + yp*PATCHSIZE);
		d = (int)floor(rnum[ofs+2]*9);
		
		x2 = x1 + (d%3)-1;
		y2 = y1 + (d/3)-1;
		
		tryMove(x1,y1,x2,y2,Grid,dN,P,rnum[ofs+3]);
	}
}
#endif

void doMCStep()
{
#ifdef CUDA
	int STEP = 2*PATCHSIZE*PATCHSIZE;
	int block_size = 32;
	int n_blocks = NTHREADS/block_size + (NTHREADS%block_size == 0 ? 0 : 1);
	
	hiprandGenerateUniform(gen, randnum, NTHREADS*RNC*STEP);
	for (int i=0;i<STEP;i++)
	{
		int ox=rand()%PATCHSIZE;
		int oy=rand()%PATCHSIZE;
		int flip=rand()%2;

		PatchKernel <<< n_blocks, block_size >>> (dGrid,ox,oy,flip,P,randnum,i,dN);
	}
	hipMemcpy(cGrid,dGrid,sizeof(int)*XR*YR,hipMemcpyDeviceToHost);
#else
	for (int i=0;i<XR*YR;i++)
	{
		int x1 = rand()%XR;
		int y1 = rand()%YR;
		int x2 = x1+rand()%3-1;
		int y2 = y1+rand()%3-1;
		
		tryMove(x1,y1,x2,y2,cGrid,dN,P,(rand()%1000001)/1000000.0);
	}
#endif
}

int drawFromEnvironment()
{
	int val = rand()%BASES;
	if (rand()%20 != 0) val = 0;
	
	return val;
}

void Init()
{
#ifdef CUDA
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	
	hipMalloc((void **)&randnum, PATCHSIZE*PATCHSIZE*2*NTHREADS*RNC*sizeof(float));
	hipMalloc((void **)&dGrid, XR*YR*sizeof(int));
	hipMalloc((void **)&dN, BASES*XR*YR*sizeof(int));
	hipMalloc((void **)&P.W, BASES*BASES*NEIGHBORS*sizeof(float));
#else
	P.W = (float*)malloc(BASES*BASES*NEIGHBORS*sizeof(float));
	dN = (int*)malloc(BASES*XR*YR*sizeof(int));
#endif

	cGrid=(int*)malloc(sizeof(int)*XR*YR);
	
	for (int i=0;i<XR*YR;i++)
	{
		cGrid[i]=drawFromEnvironment();
	}
	
#ifdef CUDA
	hipMemcpy(dGrid,cGrid,sizeof(int)*XR*YR,hipMemcpyHostToDevice);
#endif
}

void Render()
{
	int r,g,b;
	int x,y,x0,y0;
	int sx = XRes/XR, sy = YRes/YR;
	int xm,ym;
	int loc;
	//~ memset(ScreenBuf,0,XRes*YRes*Bpp);
	
	for (y=0;y<YR;y++)
	{
		for (x=0;x<XR;x++)
		{
			x0 = x*sx; y0 = y*sy;
			loc = cGrid[x+y*XR];
			int sr = ScreenBuf[(x0+y0*XRes)*Bpp+0];
			int sg = ScreenBuf[(x0+y0*XRes)*Bpp+1];
			int sb = ScreenBuf[(x0+y0*XRes)*Bpp+2];
			
			r = (sr<colors[loc*3+0])*colors[loc*3+0]/20 + sr;
			g = (sg<colors[loc*3+1])*colors[loc*3+1]/20 + sg;
			b = (sb<colors[loc*3+2])*colors[loc*3+2]/20 + sb;
			
			r = min(r,255); g = min(g,255); b = min(b,255);
			
			for (ym=y0;ym<y0+sy;ym++)
				for (xm=x0;xm<x0+sx;xm++)
				{
					ScreenBuf[(xm+ym*XRes)*Bpp+0]=r;
					ScreenBuf[(xm+ym*XRes)*Bpp+1]=g;
					ScreenBuf[(xm+ym*XRes)*Bpp+2]=b;
				}
		}
	}
}

void doOcean()
{
	for (int y=0;y<30;y++)
	{		
		for (int x=0;x<XR;x++)
		{
			cGrid[x+y*XR]=drawFromEnvironment();
		}
	}
}

void doShift()
{
	for (int y=0;y<30;y++)
	{		
		for (int x=XR-1;x>0;x--)
		{
			cGrid[x+y*XR]=cGrid[x-1+y*XR];
		}
		cGrid[0+y*XR] = drawFromEnvironment();
	}
}

void doVShift()
{
	for (int x=0;x<XR;x++)
	{
		for (int y=YR-1;y>30;y--)
		{		
			cGrid[x+y*XR]=cGrid[x+(y-1)*XR];
		}
		cGrid[x+30*XR] = 0; //drawFromEnvironment();
	}
}

int main()
{
	char Str[512];
	Img I;
	int lastf=clock();
	
	srand(time(NULL));
	
	XRes=XR*4;
	YRes=YR*4;
	
	Bpp=4;
	ScreenBuf=(unsigned char*)malloc(XRes*YRes*Bpp);
	I.Width=XRes; I.Height=YRes;
	I.Image=ScreenBuf;
	InitSDL();
	
	Init();
	setParams();
	printf("Initialized\n");

	setNeighbors();
	int frame=0;
	
	while (1)
	{
		doMCStep();
		Render();
		
		if (frame%50 == 0)
		{
			FILE *f=fopen("hist.txt","a");
			int counts[BASES];
			memset(counts,0,sizeof(int)*BASES);
			for (int y=30;y<YR;y++)
				for (int x=0;x<XR;x++)				
					counts[cGrid[x+y*XR]]++;
			
			for (int i=0;i<BASES;i++)
				fprintf(f,"%d ",counts[i]);
			
			fprintf(f,"\n");
			fclose(f);
			
			PNMSave("tmp.pnm",I);
			sprintf(Str,"pnmtopng -force tmp.pnm > frames/%.6d.png", frame/10);
			system(Str);
			BlitBuf(ScreenBuf,0,0,XRes,YRes);
			for (int i=0;i<XRes*YRes*Bpp;i++)
				ScreenBuf[i]/=2.0;
		}
		frame++;
		
		doShift();
		//~ if (frame%150 == 0)
		//~ {
			//~ doVShift();
		//~ }
				
#ifdef CUDA
		hipMemcpy(dGrid,cGrid,sizeof(int)*XR*YR,hipMemcpyHostToDevice);
#endif
		setNeighbors();

		printf("%d\n",clock()-lastf);
		lastf=clock();
		
		int Ch=ReadKey();		
		if (Ch=='q') return 0;
		if (Ch=='p') setParams();
	}   
}
